#include "hip/hip_runtime.h"
<<<<<<< HEAD
/* compile with nvcc -arch=sm_35 -o tester jacobi_f.o jacobi_cpu.o */

=======
>>>>>>> a2c9a02f5f6faa572ad9c322fd3779207cb2faec
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "jacobi_f.h"
#include "jacobi_cpu.h"

int main(int argc, char *argv[])
{

	if(argc != 2)
	{
		printf("not enough arguments. supply matrix size.");
		exit(1);
	}

	int size = atoi(argv[1]);
	
	double* M = (double*)malloc(size*size*sizeof(double));
	int row, col;
	for (row = 0; row < size; row++)
		for (col = 0; col <= row; col++)
			*(M+row*size+col) = (double)rand()/(double)RAND_MAX*100;
	
	for (row = 0; row < size; row++)
		for (col = 0; col < (size - row - 1); col++)
			*(M+col*size+row) = *(M+row*size+col);

	printf("For matrix size %d*%d\n", size, size);

<<<<<<< HEAD
	/* recoding time for serial version*/
=======
>>>>>>> a2c9a02f5f6faa572ad9c322fd3779207cb2faec
	clock_t start = clock(), diff;
	jacobi_c(M,size);
	diff = clock() - start;

	double t_in_sec = (double)diff/(double)CLOCKS_PER_SEC;
	printf("Time taken for CPU jacobi: %f seconds.\n", t_in_sec);
	
<<<<<<< HEAD
	/* recording time for parallel version */
=======
>>>>>>> a2c9a02f5f6faa572ad9c322fd3779207cb2faec
	start = clock();
	jacobi_cu(M,size);
	diff = clock() - start;

	t_in_sec = (double)diff/(double)CLOCKS_PER_SEC;
	printf("Time taken for GPU jacobi: %f seconds.\n", t_in_sec);

	return 0;
<<<<<<< HEAD
}
=======
}
>>>>>>> a2c9a02f5f6faa572ad9c322fd3779207cb2faec
