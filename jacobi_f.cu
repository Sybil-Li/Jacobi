#include "hip/hip_runtime.h"
/*
real	0m1.637s
user	0m1.142s
sys		0m0.463s 

real	0m1.528s
user	0m0.931s
sys		0m0.561s
*/

#include "jacobi_f.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


__global__ void check (double *dev_A, int n, double tolerance, int *d_cont/*, int *d_indicator*/);
__global__ void jacobi(double *dev_A, double *dev_V, int *dev_pair, int size);

void jacobi_cu( double* M, int n)
{

	int row, col, i;	
	double tolerance = 0.000000000001;
	int *d_cont, cont = 1;
	hipMalloc((void**) &d_cont, sizeof(int));
	hipMemcpy(d_cont, &cont, sizeof(int), hipMemcpyHostToDevice);

	double* A = M;

	double* V = (double*)malloc(1024*1024*sizeof(double));
	int* pair = (int*)malloc(n*sizeof(int));

	double *d_A;
	double *d_V;
	int *d_pair;
	//int *d_indicator;
	hipMalloc( (void**) &d_A, 1024*1024*sizeof(double));
	hipMalloc( (void**) &d_V, 1024*1024*sizeof(double));
	hipMalloc( (void**) &d_pair, n*sizeof(int));
	//hipMalloc( (void**) &d_indicator, 1024*1024*sizeof(int));

	

	/*copy matrix to device*/
	hipMemcpy(d_A, A, 1024*1024*sizeof(double), hipMemcpyHostToDevice);

	/*initializing vector matrix V */
	for (row = 0; row < n; row++) 
	{
		for (col = 0; col < n; col++) 
		{
			if (row == col)
			{
				*(V + row * n + col) = 1.0;
				//*(indicator + row * n) = 0;
			}
			else
			{
				*(V + row * n + col) = 0.0;
				//*(indicator + row * n) = 0;
			}
		}
	}

	/*copy matrix to device*/
	//hipMemcpy(d_V, V, 1024*1024*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(d_indicator, indicator, 1024*1024*sizeof(int), hipMemcpyHostToDevice);


	/*initializing pair matrix*/
	for (i = 0; i < n; i++)
		*(pair + i) = i;

	//for (i = 0; i < n; i++)
		//printf("%d ", *(pair + i));

	/*copy matrix to device*/
	hipMemcpy(d_pair, pair, n*sizeof(int), hipMemcpyHostToDevice);

	/*launch kernel here*/
	dim3 grid (1, 1, 1);
	dim3 block (n/2, 1, 1);

	int iteration = 0;
	while ((cont != 0) && (iteration <= 100000))
	{
		jacobi<<<grid, block>>>(d_A, d_V, d_pair, n);
		cont = 0;
		hipMemcpy(d_cont, &cont, sizeof(int), hipMemcpyHostToDevice);
		check<<<16, dim3(n/16, 1, 1)>>>(d_A, n, tolerance, d_cont/*,d_indicator*/);
		hipMemcpy(&cont, d_cont, sizeof(int), hipMemcpyDeviceToHost);
		/*hipMemcpy(indicator, d_indicator, 1024*1024*sizeof(int), hipMemcpyDeviceToHost);
		for (row = 0; row<n; row++) {
			for (col = 0; col<n; col++)
				printf("%d ", *(indicator+row*n+col));
			printf("\n");
		}*/

		iteration++;
		
	}
	
	hipMemcpy(pair, d_pair, n*sizeof(int), hipMemcpyDeviceToHost);
	//for (int i = 0; i < n; i++)
		//printf("%d\n", *(pair + n));

	/*write matrix back to host*/
	hipMemcpy(A, d_A, 1024*1024*sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(V, d_V, 1024*1024*sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(indicator, d_indicator, 1024*1024*sizeof(int), hipMemcpyDeviceToHost);
	/*for (row = 0; row<n; row++) {
		for (col = 0; col<n; col++)
			printf("%d ", *(indicator+row*n+col));
		printf("\n");
	}*/

	/*check result*/
	double* ans = (double*) malloc(n*sizeof(double));
	//double norm = 0;

	for (row = 0; row<n; row++){
		for (col = 0; col<n; col++){
			if (row==col)
			{
				//*(ans+row) = *(A+row*n+col);
				//norm += (*(ans+row))*(*(ans+col));
				printf("%lf\n", *(A+row*n+col));
			}
			
			//printf("%lf", *(A+row*n+col));
		}
		//printf("\n");
	}
	//norm = sqrt(norm);
	//printf("Norm is %lf\n", norm);

	free(A);
	free(V);
	free(pair);
	hipFree(d_A);
	hipFree(d_V);
	hipFree(d_pair);

}

__global__ void jacobi(double *dev_A, double *dev_V, int *dev_pair, int size)
{
	short threadno, p, q, n, i, temp1, temp2;
	double c, s;
	threadno = threadIdx.x;
	n = size;	

	p = *(dev_pair + threadno);
	q = *(dev_pair + threadno + n/2);


	/*calculate c, s value*/
	if (*(dev_A + p * n + q) != 0)
	{
		double torque, t;
		torque = ( *(dev_A + q * n + q) - *(dev_A + p * n + p))/(2*(*(dev_A + p * n + q)));
		if (torque >= 0)
		    t = 1/(torque + sqrt(1+torque*torque));
		else
		    t = -1/(-torque + sqrt(1+torque*torque));
		
		c = 1/sqrt(1+t*t);
		s = t*c;
	}
	else
	{
		c = 1;
		s = 0;
	}

	/* A = transpose(J)*A*J */
	for (i = 0; i < n; i++)
	{
		double Api = (*(dev_A + p * n + i))*c + (*(dev_A + q * n + i))*(-s);
		double Aqi = (*(dev_A + p * n + i))*s + (*(dev_A + q * n + i))*c;
		__syncthreads();
		*(dev_A + p * n + i) = Api;
		*(dev_A + q * n + i) = Aqi;
	}


	for (i = 0; i < n; i++)
	{ 
		double Aip = (*(dev_A + i * n + p))*c + (*(dev_A + i * n + q))*(-s);
		double Aiq = (*(dev_A + i * n + p))*s + (*(dev_A + i * n + q))*c;
		__syncthreads();
		*(dev_A + i * n + p) = Aip;
		*(dev_A + i * n + q) = Aiq;
	}
	 

	/* V = V*J */
	/*for (i = 0; i < n; i++)
	{ 
		double Vpi = (*(dev_V + p * n + i))*c + (*(dev_V + q * n + i))*(-s);
		double Vqi = (*(dev_V + p * n + i))*s + (*(dev_V + q * n + i))*c;
		__syncthreads();
		*(dev_V + p * n + i) = Vpi;
		*(dev_V + q * n + i) = Vqi;
	}*/

	/* chess tournament rotate*/
	if (threadno == 0)	
	{
		temp1 = 0;
		temp2 = *(dev_pair + n/2 + 1);
	}
	else if (threadno == 1)
	{
		temp1 = *(dev_pair + n/2);
		temp2 = *(dev_pair + threadno + n/2 + 1);
	}
	else if (threadno == n/2 - 1)
	{
		temp1 = *(dev_pair + threadno - 1);
		temp2 = *(dev_pair + n/2 - 1);
	}
	else
	{
		temp1 = *(dev_pair + threadno - 1);
		temp2 = *(dev_pair + threadno + n/2 + 1);
	}

	__syncthreads();

	*(dev_pair + threadno) = temp1;
	*(dev_pair + threadno + n/2) = temp2;
	
}

__global__ void check (double *dev_A, int n, double tolerance, int *d_cont/*, int *d_indicator*/)
{
	int threadno = blockIdx.x * n/16 + threadIdx.x;
	for (int i = 0; i < n; i++)
	{	
		if (threadno != i)
		{
			if (*(dev_A + threadno * n + i) > tolerance)
			{
				//*(d_indicator + threadno * n + i) = 1;
				*d_cont = 1;
			}
			//else
				//*(d_indicator + threadno * n + i) = 0;
		} 
	}
}






		

